#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

#include <shoverand/prng/mrg32k3a/MRG32k3a.hxx>
#include <shoverand/core/RNG.hxx>

#include "dummy_generator/dummy_generator.hxx"

using shoverand::RNG;
using shoverand::MRG32k3a;

typedef float float_type;
//typedef RNG < float_type, MRG32k3a > random_engine_type;
typedef RNG < float_type, DummyGenerator > random_engine_type;

__global__ void kernelMonteCarloPi( float_type * outArray )
{

   random_engine_type    rng;
   float_type x, y;
   long pointsInsideCircle = 0;
   int indiceThd     = threadIdx.x;  // thread identifier

   for (int i = 0; i < 5000; i++)
   {
       x = rng.next();
       y = rng.next();

       if (sqrt (x*x + y*y) <= 1.0) {
           pointsInsideCircle += 1;
       }
   }

   outArray[indiceThd] = (pointsInsideCircle * 4.0) / 5000.0;
}

int main( int argc, char * argv[] )
{
   const int   arraySize  = 200;   // array size

   float_type *     array_host;           // array on the host
   float_type *     array_device;         // array on the device
   float_type       estimated_pi = 0.0;

   hipError_t error;             // error code 

   // memory allocation for the host array
   array_host    = new float_type[ arraySize ];
   // memory allocation for the device array
   error = hipMalloc( (void **) &array_device, arraySize * sizeof( float_type ) );
   std::cout << "Error during memory allocation: " << hipGetErrorString(error) << std::endl;

   random_engine_type ::init(arraySize);

   // -------------------------------------------//
   // Kernel call (1 block, arraySize threads)  //
   // ----------------------------------------- //
   kernelMonteCarloPi<<< 1, arraySize >>>( array_device );
   error = hipGetLastError();
   std::cout << "Error during kernel call: " << hipGetErrorString(error) << std::endl;

   // copy back resulting data from device's array to host's
   error = hipMemcpy( array_host, array_device, arraySize * sizeof( float_type ), hipMemcpyDeviceToHost );
   std::cout << "Error in copy from device to host: " << hipGetErrorString(error) << std::endl;

   std::cout << "Received results" << std::endl;
   for ( int i = 0; i < arraySize; ++i )
   {
      std::cout << array_host[ i ] << " ";
      estimated_pi += array_host[ i ];
   }
   std::cout << std::endl;

   estimated_pi /= (float_type)arraySize;
   std::cout << "Estimated PI " << estimated_pi << std::endl;

   random_engine_type::release();

   // free device's memory
   hipFree( array_device );
   // free host's memory
   delete array_host;

   return 0;
}

