#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

#include "dummy_generator/dummy_generator_original.hxx"

typedef float float_type;

__global__ void kernelMonteCarloPi( float_type * outArray )
{

   DummyGenerator<float_type>    rng;
   float_type x, y;
   long pointsInsideCircle = 0;
   int indiceThd     = threadIdx.x;  // thread identifier

   for (int i = 0; i < 5000; i++)
   {
       x = rng.next();
       y = rng.next();

       if (sqrt (x*x + y*y) <= 1.0) {
           pointsInsideCircle += 1;
       }
   }

   outArray[indiceThd] = (pointsInsideCircle * 4.0) / 5000.0;
}

int main( int argc, char * argv[] )
{
   const int   arraySize  = 200;   // array size

   float_type *     array_host;           // array on the host
   float_type *     array_device;         // array on the device
   float_type       estimated_pi = 0.0;

   hipError_t error;             // error code 

   // memory allocation for the host array
   array_host    = new float_type[ arraySize ];
   // memory allocation for the device array
   error = hipMalloc( (void **) &array_device, arraySize * sizeof( float_type ) );
   std::cout << "Error during memory allocation: " << hipGetErrorString(error) << std::endl;

   // -------------------------------------------//
   // Kernel call (1 block, arraySize threads)  //
   // ----------------------------------------- //
   kernelMonteCarloPi<<< 1, arraySize >>>( array_device );
   error = hipGetLastError();
   std::cout << "Error during kernel call: " << hipGetErrorString(error) << std::endl;

   // copy back resulting data from device's array to host's
   error = hipMemcpy( array_host, array_device, arraySize * sizeof( float_type ), hipMemcpyDeviceToHost );
   std::cout << "Error in copy from device to host: " << hipGetErrorString(error) << std::endl;

   std::cout << "Received results" << std::endl;
   for ( int i = 0; i < arraySize; ++i )
   {
      std::cout << array_host[ i ] << " ";
      estimated_pi += array_host[ i ];
   }
   std::cout << std::endl;

   estimated_pi /= (float_type)arraySize;
   std::cout << "Estimated PI " << estimated_pi << std::endl;

   // free device's memory
   hipFree( array_device );
   // free host's memory
   delete array_host;

   return 0;
}

