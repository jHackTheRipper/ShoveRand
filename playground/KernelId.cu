#include <hip/hip_runtime.h>
#include <iostream>

#define ARRAY_SIZE (64 * 64)

__global__
void myKernel2 (uint* array) {
   uint id;

   asm("mov.u32 %0, %gridid;" : "=r"(id) );

   array [ threadIdx.x + blockIdx.x * blockDim.x ] = id;
}

__global__
void myKernel (uint* array) {
   uint id;

   asm("mov.u32 %0, %gridid;" : "=r"(id) );

   array [ threadIdx.x + blockIdx.x * blockDim.x ] = id;
}

int main() {
   uint array[ARRAY_SIZE] = { 0 };
   uint* d_array;

   hipMalloc( (void**) &d_array, sizeof(uint) * ARRAY_SIZE);

   for (int j = 0; j < 10; ++j) {
      myKernel <<< 64, 64 >>> ( d_array );
      myKernel2 <<< 64, 64 >>> ( d_array );

      hipMemcpy( array, d_array, sizeof(uint) * ARRAY_SIZE, hipMemcpyDeviceToHost);

      for (int i = 0; i < ARRAY_SIZE; ++i) {
         std::cout << array[i] << "\n";
      }
      std::cout << std::endl;
   }
   hipFree (d_array);
   return 0;
}

