
#include <hip/hip_runtime.h>
/** Empty since function/methods calls implies inlining.
 \see SubStream.h
 */


/*#include <cuda.h>

#include <SubStream.h>
#include <Stream.h>
#include <utils.h>
*/

namespace MRG32k3a {
 
   /** Init through RngStream
    \param s Stream used as a base for current sub-stream
    \author jH@CKtheRipper
    */
  /* __device__
   SubStream::SubStream(Stream allStreams[]) {
      // get stream corresponding to block id
      Stream* stream  = allStreams + (blockIdx.x * blockDim.x + blockIdx.y);
   
      // copy stream state in current SubStream
      for (unsigned i = 0; i < 6; ++i) {
         Cg_[i] = stream->getCg(i);
         Bg_[i] = stream->getBg(i);
      }
      
      // advance to the SubStream corresponding to thread id
      this->jumpAhead(threadIdx.x + blockDim.x * threadIdx.y);
   }
   
   */
   
   /** 
    Draw next number
    \author jH@CKtheRipper
    */
 /*  __device__
   double SubStream::next() {
      long k;
      double p1, p2, u;
      
      // changed static_casts to regular casts due to cuda
      
      // Component 1
      p1 = a12 * Cg_[1] - a13n * Cg_[0];
      k = (long) (p1 / m1);
      p1 -= k * m1;
      
      if (p1 < 0.0)  p1 += m1;
      
      Cg_[0] = Cg_[1]; Cg_[1] = Cg_[2]; Cg_[2] = p1;
      
      // Component 2
      p2 = a21 * Cg_[5] - a23n * Cg_[3];
      k = (long) (p2 / m2);
      p2 -= k * m2;
      
      if (p2 < 0.0) p2 += m2;
      
      Cg_[3] = Cg_[4]; Cg_[4] = Cg_[5]; Cg_[5] = p2;
      
      // Combination
      u = ((p1 > p2) ? (p1 - p2) * norm : (p1 - p2 + m1) * norm);
      
      return u;
   }
  */ 
   
   /** 
    Rise A matrices to the power of n.
    
    \param n Number of jumps through the SubStreams.
    If n is set to 0, the current stream is 
    resetted to its original state.
    \author jH@CKtheRipper
    */
   /* __device__ 
    void SubStream::jumpAhead(long pow) {
      double A1_pN[3][3] = {
         {       0.0,        1.0,       0.0 },
         {       0.0,        0.0,       1.0 },
         { -810728.0,  1403580.0,       0.0 }
      };
      
      double A2_pN[3][3] = {
         {        0.0,        1.0,       0.0 },
         {        0.0,        0.0,       1.0 },
         { -1370589.0,        0.0,  527612.0 }
      };
      
      
      
      MyMatPowModM(A1_pN, A2_pN, pow);
      
      
      MatVecModM(A1_pN, Bg_, Bg_, m1);
      MatVecModM(A2_pN, &Bg_[3], &Bg_[3], m2);
      
      for (unsigned i = 0; i < 6; ++i) {
         Cg_[i] = Bg_[i];
      }
   }
   */
} // end of namespace MRG32k3a
