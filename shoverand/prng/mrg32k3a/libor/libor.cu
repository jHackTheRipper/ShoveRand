#include "hip/hip_runtime.h"

/* Program to compute swaption portfolio using NVIDIA CUDA */

#include <stdio.h>
#include <cutil_inline.h>

// parameters for LIBOR calculation

#include <params.h>

////////////////////////////////////////////////////////////////////////
// include kernel routines
////////////////////////////////////////////////////////////////////////

#include <mrg32k3a_cuda.h>
#include <libor_kernels.cu>

////////////////////////////////////////////////////////////////////////
// declaration of CPU calls
////////////////////////////////////////////////////////////////////////

extern "C"
void copy_constants(int, int, int, int *, float, float *, float *, float *);

extern "C"
void Pathcalc_Portfolio_CPU(float *, float *, float *);

extern "C"
void Pathcalc_Portfolio_CPU2(float *, float *);

extern "C" 
void CPU_mrg32k3a_init(uint *V1, uint *V2, int offset);

extern "C" 
void CPU_mrg32k3a_normal(int, float *);


////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv){
    
  int    N=NN, Nmat=NMAT, Nopt=NOPT, 
         maturities[] = {4,4,4,8,8,8,20,20,20,28,28,28,40,40,40};
  float  delta=0.25, lambda[NN], L0[NN],
         swaprates[]  = {.045,.05,.055,.045,.05,.055,.045,.05,
                         .055,.045,.05,.055,.045,.05,.055 };

  uint  V1[] = { 1, 2, 3 };  // seeds
  uint  V2[] = { 1, 2, 3 };  // seeds

  // 'h_' prefix - CPU (host) memory space

  int     i;
  float  *h_v, *h_Lb, *h_v2, *h_Lb2, *h_z;
  double  v, v_err, Lb, Lb_err; 

  unsigned int hTimer;
  double  gpuTime;

  // 'd_' prefix - GPU (device) memory space

  float  *d_v, *d_Lb, *d_z;

  CUT_DEVICE_INIT(argc, argv);
  CUT_SAFE_CALL( cutCreateTimer(&hTimer) );

  // Copy all constants into device constant memory

  for (i=0; i<NN; i++) lambda[i] = 0.2;
  for (i=0; i<NN; i++) L0[i] = 0.051;

  hipMemcpyToSymbol(HIP_SYMBOL("N"), &N, sizeof(N));
  hipMemcpyToSymbol(HIP_SYMBOL("Nmat"), &Nmat, sizeof(Nmat));
  hipMemcpyToSymbol(HIP_SYMBOL("Nopt"), &Nopt, sizeof(Nopt));
  hipMemcpyToSymbol(HIP_SYMBOL("maturities"), maturities, sizeof(maturities));
  hipMemcpyToSymbol(HIP_SYMBOL("delta"), &delta, sizeof(delta));
  hipMemcpyToSymbol(HIP_SYMBOL("lambda"), lambda, sizeof(lambda));        
  hipMemcpyToSymbol(HIP_SYMBOL("L0"), L0, sizeof(L0));        
  hipMemcpyToSymbol(HIP_SYMBOL("swaprates"), swaprates, sizeof(swaprates));

  copy_constants(N,Nmat,Nopt,maturities,delta,lambda,L0,swaprates);

  // Allocate memory on host and device

  h_v      = (float *)malloc(sizeof(float)*NPATH);
  h_v2     = (float *)malloc(sizeof(float)*NPATH);
  CUDA_SAFE_CALL( hipMalloc((void **)&d_v, sizeof(float)*NPATH) );
  h_Lb     = (float *)malloc(sizeof(float)*NPATH);
  h_Lb2    = (float *)malloc(sizeof(float)*NPATH);
  CUDA_SAFE_CALL( hipMalloc((void **)&d_Lb, sizeof(float)*NPATH) );
  h_z      = (float *)malloc(sizeof(float)*NPATH*Nmat);
  CUDA_SAFE_CALL( hipMalloc((void **)&d_z, sizeof(float)*NPATH*Nmat) );

  // Random number generation

  gpu_mrg32k3a_init(V1, V2, 0);
  gpu_mrg32k3a_normal(NPATH/64, 64, Nmat, d_z);

  CPU_mrg32k3a_init(V1, V2, 0);
  CPU_mrg32k3a_normal(NPATH*Nmat, h_z);

  // CUDA computation on GPU

  CUT_SAFE_CALL( cutResetTimer(hTimer) );
  CUT_SAFE_CALL( cutStartTimer(hTimer) );
  Pathcalc_Portfolio_GPU2<<<NPATH/64, 64>>>(d_z, d_v);
  CUT_CHECK_ERROR("Pathcalc_Portfolio_GPU2() execution failed\n");
  CUDA_SAFE_CALL( hipDeviceSynchronize() );
  CUT_SAFE_CALL( cutStopTimer(hTimer) );

  gpuTime = cutGetTimerValue(hTimer);
  printf("\nGPU time (No Greeks) : %f msec\n", gpuTime);

  // Gold computation on CPU

  CUT_SAFE_CALL( cutResetTimer(hTimer) );
  CUT_SAFE_CALL( cutStartTimer(hTimer) );
  Pathcalc_Portfolio_CPU2(h_z, h_v2);
  CUT_SAFE_CALL( cutStopTimer(hTimer) );

  gpuTime = cutGetTimerValue(hTimer);
  printf("CPU time (No Greeks) : %f msec\n", gpuTime);

  // fetch CUDA results and compare to Gold results

  CUDA_SAFE_CALL( hipMemcpy(h_v, d_v, sizeof(float)*NPATH,
                  hipMemcpyDeviceToHost) );

  v     = 0.0;
  v_err = 0.0;
  for (i=0; i<NPATH; i++) {
    v     += h_v[i] / NPATH;
    v_err += abs(h_v[i]-h_v2[i]) / NPATH;
  }

  printf("average value v  = %15.8f\n", v);
  printf("average error    = %15.8f\n", v_err);

  // CUDA computation on GPU

  CUT_SAFE_CALL( cutResetTimer(hTimer) );
  CUT_SAFE_CALL( cutStartTimer(hTimer) ); 
  Pathcalc_Portfolio_GPU<<<NPATH/64, 64>>>(d_z, d_v,d_Lb);
  CUT_CHECK_ERROR("Pathcalc_Portfolio_GPU() execution failed\n");
  CUDA_SAFE_CALL( hipDeviceSynchronize() );
  CUT_SAFE_CALL( cutStopTimer(hTimer) );

  gpuTime = cutGetTimerValue(hTimer);
  printf("\nGPU time (Greeks) : %f msec\n", gpuTime);

  // Gold computation on CPU

  CUT_SAFE_CALL( cutResetTimer(hTimer) );
  CUT_SAFE_CALL( cutStartTimer(hTimer) );
  Pathcalc_Portfolio_CPU(h_z, h_v2,h_Lb2);
  CUT_SAFE_CALL( cutStopTimer(hTimer) );

  gpuTime = cutGetTimerValue(hTimer);
  printf("CPU time (Greeks) : %f msec\n", gpuTime);

  // fetch CUDA results and compare to Gold results

  CUDA_SAFE_CALL( hipMemcpy(h_v, d_v, sizeof(float)*NPATH,
                  hipMemcpyDeviceToHost) );
  CUDA_SAFE_CALL( hipMemcpy(h_Lb, d_Lb, sizeof(float)*NPATH,
                  hipMemcpyDeviceToHost) );
    
  v      = 0.0;
  v_err  = 0.0;
  Lb     = 0.0;
  Lb_err = 0.0;
  for (i=0; i<NPATH; i++) {
    v      += h_v[i] / NPATH;
    v_err  += abs(h_v[i]-h_v2[i]) / NPATH;
    Lb     += h_Lb[i] / NPATH;
    Lb_err += abs(h_Lb[i]-h_Lb2[i]) / NPATH;
  }

  printf("average value v  = %15.8f\n", v);
  printf("average error    = %15.8f\n", v_err);
  printf("average value Lb = %15.8f\n", Lb);
  printf("average error    = %15.8f\n", Lb_err);

  // Release memory and exit cleanly

  free(h_v);
  free(h_v2);
  free(h_Lb);
  free(h_Lb2);

  CUDA_SAFE_CALL( hipFree(d_v));
  CUDA_SAFE_CALL( hipFree(d_Lb));

  CUT_SAFE_CALL( cutDeleteTimer(hTimer) );
  CUT_EXIT(argc, argv);
}
