#include "hip/hip_runtime.h"
#include <SubStream.h>
#include <Stream.h>

#include <cstdio>
#include <cstdlib>
#include <cerrno>


#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cutil_inline_runtime.h>


__global__ void testMRG32k3a(double* ddata, MRG32k3a::Stream allStreams[]) {
   
  // MRG32k3a::SubStream s(allStreams);4
  // __syncthreads();
   
   
   ddata[blockDim.x * blockIdx.x + threadIdx.x] = 23;
   __syncthreads();
}


int main(int, char **) {

   int block_num = 1;
   int thread_num = 32;
   int data_size = block_num * thread_num * sizeof(double);
   
   double* d_data;
   double* h_data;
   hipError_t e;
   float gputime;
   
   
   // create Streams for all threads
   // MRG32k3a::Stream* allStreamslocal = new MRG32k3a::Stream[block_num];
   //MRG32k3a::Stream::init(block_num);
   
   // create timers 
   hipEvent_t start;
   hipEvent_t stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   
   
   // allocate memory for data on device
   cutilSafeCall( hipMalloc((void**) &d_data, data_size) );
   cutilSafeCall( hipMemset(d_data, 42, data_size) );
   
   if (hipGetLastError() != hipSuccess) {
      fprintf(stderr, "error has occured before kernel call.\n");
      exit(1);
   }
   
   hipEventRecord(start, 0);
   
   // kernel call
   testMRG32k3a<<< block_num, thread_num>>>(d_data, MRG32k3a::Stream::allStreams);
   
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   
   e = hipGetLastError();
   if (e != hipSuccess) {
      fprintf(stderr, "failure in kernel call.\n%s\n", hipGetErrorString(e));
      exit(2);
   }
   
   // allocate memory to get results back on the host
   h_data = (double *) malloc(data_size);
   
   if (h_data == NULL) {
      fprintf(stderr, "failure in allocating host memory for output data.\n");
      exit(3);
   }
   
   cutilSafeCall(
                 hipMemcpy(h_data,
                            d_data,
                            data_size,
                            hipMemcpyDeviceToHost));
   hipEventElapsedTime(&gputime, start, stop);
   
   
   for (int i = 0; i < block_num * thread_num; ++i) {
      printf ("h_data[%d] = %lf\n", i, h_data[i]);
   }
   
   
   printf("generated numbers: %d\n", thread_num * block_num);
   printf("Processing time: %f (ms)\n", gputime);
   printf("Samples per second: %E \n", (thread_num * block_num) / (gputime * 0.001));
   
   //free memories
   hipEventDestroy(start);
   hipEventDestroy(stop);
   
   free(h_data);
   cutilSafeCall(hipFree(d_data));
   
}
