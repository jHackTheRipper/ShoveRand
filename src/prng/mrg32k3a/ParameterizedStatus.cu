
#include "ParameterizedStatus.h"
#include "Stream.h"
 
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cutil_inline_runtime.h>

	
/******** Host equivalents to device constants *********/
const double ParameterizedStatus<shoverand::prng::MRG32k3a::MRG32k3a> ::  A1p0_host[3][3] = {
	{       0.0,        1.0,       0.0 },
	{       0.0,        0.0,       1.0 },
	{ -810728.0,  1403580.0,       0.0 } 
};

const double ParameterizedStatus<shoverand::prng::MRG32k3a::MRG32k3a> ::  A2p0_host[3][3] = {
	{        0.0,        1.0,       0.0 },
	{        0.0,        0.0,       1.0 },
	{ -1370589.0,        0.0,  527612.0 }
};

const double ParameterizedStatus<shoverand::prng::MRG32k3a::MRG32k3a> ::  A1p76_host[3][3] = {
	{      82758667.0, 1871391091.0, 4127413238.0 },
	{    3672831523.0,   69195019.0, 1871391091.0 },
	{    3672091415.0, 3528743235.0,   69195019.0 }
};

const double ParameterizedStatus<shoverand::prng::MRG32k3a::MRG32k3a> ::  A2p76_host[3][3] = {
	{    1511326704.0, 3759209742.0, 1610795712.0 },
	{    4292754251.0, 1511326704.0, 3889917532.0 },
	{    3859662829.0, 4292754251.0, 3708466080.0 }
};


const double ParameterizedStatus<shoverand::prng::MRG32k3a::MRG32k3a> :: A1p127_host[3][3] = {
	{    2427906178.0, 3580155704.0,  949770784.0 },
	{     226153695.0, 1230515664.0, 3580155704.0 },
	{    1988835001.0,  986791581.0, 1230515664.0 }
};

const double ParameterizedStatus<shoverand::prng::MRG32k3a::MRG32k3a> :: A2p127_host[3][3] = {
	{    1464411153.0,  277697599.0, 1610723613.0 },
	{      32183930.0, 1464411153.0, 1022607788.0 },
	{    2824425944.0,   32183930.0, 2093834863.0 }
};

const double ParameterizedStatus<shoverand::prng::MRG32k3a::MRG32k3a> ::  A1_pows_host [11][3][3] = {
	{  { 1.0, 0.0, 0.0 }, { 0.0, 1.0, 0.0 }, { 0.0, 0.0, 1.0 } },
	{  { 82758667.0, 1871391091.0, 4127413238.0 }, { 3672831523.0,   69195019.0, 1871391091.0 }, { 3672091415.0, 3528743235.0, 69195019.0 } },
	{  { 3.36137e+09, 2.3293e+09, 9.96519e+07 } , { 2.00867e+09, 2.93176e+09, 2.3293e+09 } , { 1.11353e+09, 2.3741e+09, 2.93176e+09 } },
	{  { 2.13871e+09, 3.21018e+09, 2.30172e+08 } , { 1.01179e+09, 3.53602e+09, 3.21018e+09 } , { 2.84722e+09, 6.20673e+08, 3.53602e+09 } },
	{  { 1.83159e+09, 1.58826e+09, 1.31433e+09 } , { 2.38599e+09, 2.50808e+09, 1.58826e+09 } , { 1.78762e+09, 6.61437e+08, 2.50808e+09 } },
	{  { 2.04282e+09, 4.01645e+08, 4.01041e+09 } , { 3.99806e+09, 2.10605e+09, 4.01645e+08 } , { 2.29685e+09, 5.3118e+08, 2.10605e+09 } },
	{  { 2.1845e+09, 2.70304e+09, 2.30761e+09 } , { 1.51704e+09, 2.8237e+09, 2.70304e+09 } , { 1.78608e+09, 6.1756e+07, 2.8237e+09 } },
	{  { 3.24248e+09, 4.21073e+09, 5.20437e+08 } , { 6.68035e+07, 3.70726e+09, 4.21073e+09 } , { 1.46074e+09, 2.92168e+09, 3.70726e+09 } },
	{  { 2.32605e+09, 4.18359e+09, 4.04901e+09 } , { 2.60453e+09, 1.45391e+09, 4.18359e+09 } , { 2.31193e+09, 1.80536e+09, 1.45391e+09 } },
	{  { 4.14073e+09, 7.52526e+08, 2.82814e+09 } , { 1.99111e+09, 2.09751e+09, 7.52526e+08 } , { 1.67327e+09, 1.60257e+09, 2.09751e+09 } },
	{  { 1.96542e+09, 3.22182e+09, 2.04071e+09 } , { 1.26818e+09, 1.82502e+09, 3.22182e+09 } , { 2.88049e+09, 1.51013e+08, 1.82502e+09 } }
};

const double ParameterizedStatus<shoverand::prng::MRG32k3a::MRG32k3a> ::  A2_pows_host [11][3][3] = {
	{  { 1.0, 0.0, 0.0 }, { 0.0, 1.0, 0.0 }, { 0.0, 0.0, 1.0 } },
	{  { 1.51133e+09, 3.75921e+09, 1.6108e+09 } , { 4.29275e+09, 1.51133e+09, 3.88992e+09 } , { 3.85966e+09, 4.29275e+09, 3.70847e+09 }},
	{  { 9.72103e+08, 9.64808e+08, 8.78036e+08 } , { 4.24855e+09, 9.72103e+08, 1.92663e+09 } , { 1.44863e+09, 4.24855e+09, 3.19611e+09 } },
	{  { 9.07239e+08, 2.92629e+09, 2.86585e+09 } , { 8.40689e+08, 9.07239e+08, 1.2492e+09 } , { 4.27877e+09, 8.40689e+08, 3.3311e+09 } },
	{  { 3.49738e+09, 3.17425e+09, 3.18251e+09 } , { 3.86482e+09, 3.49738e+09, 3.0384e+09 } , { 2.54688e+09, 3.86482e+09, 2.98021e+09 } },
	{  { 7.84581e+08, 2.65342e+09, 3.8428e+08 } , { 3.46234e+09, 7.84581e+08, 1.92303e+09 } , { 3.69099e+07, 3.46234e+09, 5.64222e+08 } },
	{  { 1.40581e+09, 4.17421e+09, 2.38421e+09 } , { 1.1278e+08, 1.40581e+09, 3.41212e+09 } , { 3.92527e+09, 1.1278e+08, 1.36504e+09 } },
	{  { 2.61947e+09, 3.13258e+06, 8.04703e+08 } , { 3.63047e+09, 2.61947e+09, 2.64523e+09 } , { 3.21677e+09, 3.63047e+09, 4.07219e+09 } },
	{  { 1.77634e+09, 1.18994e+09, 4.09576e+09 } , { 3.8136e+09, 1.77634e+09, 7.89476e+08 } , { 4.1197e+09, 3.8136e+09, 2.14536e+09 } },
	{  { 3.76949e+09, 1.0387e+09, 2.72404e+09 } , { 2.82675e+08, 3.76949e+09, 8.45493e+07 } , { 3.83672e+09, 2.82675e+08, 1.4121e+09 } },
	{  { 3.41941e+08, 2.19094e+09, 3.28734e+09 } , { 4.80943e+08, 3.41941e+08, 2.6028e+09 } , { 8.53742e+08, 4.80943e+08, 4.22841e+09 } }
};


// shoverand::prng::MRG32k3a::Stream* ParameterizedStatus<shoverand::prng::MRG32k3a::MRG32k3a> :: allStreams__;
// 	
// double* ParameterizedStatus<shoverand::prng::MRG32k3a::MRG32k3a> :: A1p76__;
// double* ParameterizedStatus<shoverand::prng::MRG32k3a::MRG32k3a> :: A2p76__;
// double* ParameterizedStatus<shoverand::prng::MRG32k3a::MRG32k3a> :: A1p127__;
// double* ParameterizedStatus<shoverand::prng::MRG32k3a::MRG32k3a> :: A2p127__;
// 
// double* ParameterizedStatus<shoverand::prng::MRG32k3a::MRG32k3a> :: A1_pows__;
// double* ParameterizedStatus<shoverand::prng::MRG32k3a::MRG32k3a> :: A2_pows__;

__host__
void ParameterizedStatus<shoverand::prng::MRG32k3a::MRG32k3a> :: setUp (unsigned short blocksNumber) {

			// create streams on the host and initiate them correctly
			// THIS MUST BE DONE SEQUENTIALLY since current state relies on previsou ones
			Stream* allStreams_host = new shoverand::prng::MRG32k3a::Stream [blocksNumber];
	
			// allocate memory for streams on the device
			cutilSafeCall( hipMalloc( (void**)&allStreams_, sizeof(shoverand::prng::MRG32k3a::Stream) * blocksNumber) );
			// transfer Streams
			cutilSafeCall( hipMemcpy( allStreams_,
												allStreams_host,
												sizeof(shoverand::prng::MRG32k3a::Stream) * blocksNumber,
												hipMemcpyHostToDevice) );

			// allocate and copy from host to device matrices
			cutilSafeCall( hipMalloc((void**) &A1p76_, sizeof(double) * 9) );
			cutilSafeCall( hipMemcpy(A1p76_, A1p76_host, sizeof(double) * 9, hipMemcpyHostToDevice ) );
			cutilSafeCall( hipMalloc((void**) &A2p76_, sizeof(double) * 9) );
			cutilSafeCall( hipMemcpy(A2p76_, A2p76_host, sizeof(double) * 9, hipMemcpyHostToDevice ) );
			cutilSafeCall( hipMalloc((void**) &A1p127_, sizeof(double) * 9) );
			cutilSafeCall( hipMemcpy(A1p127_, A1p127_host, sizeof(double) * 9, hipMemcpyHostToDevice ) );
			cutilSafeCall( hipMalloc((void**) &A2p127_, sizeof(double) * 9) );
			cutilSafeCall( hipMemcpy(A2p127_, A2p127_host, sizeof(double) * 9, hipMemcpyHostToDevice ) );
			
			cutilSafeCall( hipMalloc((void**) &A1_pows_, sizeof(double) * 11 * 3 * 3) );
			cutilSafeCall( hipMemcpy(A1_pows_, A1_pows_host, sizeof(double) * 11 * 3 * 3, hipMemcpyHostToDevice ) );
			cutilSafeCall( hipMalloc((void**) &A2_pows_, sizeof(double) * 11 * 3 * 3) );
			cutilSafeCall( hipMemcpy(A2_pows_, A2_pows_host, sizeof(double) * 11 * 3 * 3, hipMemcpyHostToDevice ) );

			delete [] allStreams_host;
}

__host__
void ParameterizedStatus<shoverand::prng::MRG32k3a::MRG32k3a> :: shutdown() {

	cutilSafeCall( hipFree(A2_pows_) );
	cutilSafeCall( hipFree(A1_pows_) );
	cutilSafeCall( hipFree(A2p127_) );
	cutilSafeCall( hipFree(A1p127_) );
	cutilSafeCall( hipFree(A2p76_) );
	cutilSafeCall( hipFree(A1p76_) );
	cutilSafeCall( hipFree(allStreams_) );
}
