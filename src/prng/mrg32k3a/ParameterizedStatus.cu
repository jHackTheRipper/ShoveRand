#include "ParameterizedStatus.h"

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cutil_inline_runtime.h>


// TODO is there a way to declare const arrays as extern?
#include "Stream.cu"

__host__
void ParameterizedStatus<MRG32k3a::MRG32k3a>::setUp (unsigned short blocksNumber) {

			// create streams on the host and initiate them correctly
			// THIS MUST BE DONE SEQUENTIALLY since current state relies on previsou ones
			allStreams_host = new MRG32k3a::Stream [blocksNumber];
	
			// allocate memory for streams on the device
			cutilSafeCall( hipMalloc( (void**)&allStreams_, sizeof(MRG32k3a::Stream) * blocksNumber) );
			// transfer Streams
			cutilSafeCall( hipMemcpy( allStreams_,
												allStreams_host,
												sizeof(MRG32k3a::Stream) * blocksNumber,
												hipMemcpyHostToDevice) );

			// allocate and copy from host to device matrices
			cutilSafeCall( hipMalloc((void**) &A1p76, sizeof(double) * 9) );
			cutilSafeCall( hipMemcpy(A1p76, MRG32k3a::A1p76_host, sizeof(double) * 9, hipMemcpyHostToDevice ) );
			cutilSafeCall( hipMalloc((void**) &A2p76, sizeof(double) * 9) );
			cutilSafeCall( hipMemcpy(A2p76, MRG32k3a::A2p76_host, sizeof(double) * 9, hipMemcpyHostToDevice ) );
			cutilSafeCall( hipMalloc((void**) &A1p127, sizeof(double) * 9) );
			cutilSafeCall( hipMemcpy(A1p127, MRG32k3a::A1p127_host, sizeof(double) * 9, hipMemcpyHostToDevice ) );
			cutilSafeCall( hipMalloc((void**) &A2p127, sizeof(double) * 9) );
			cutilSafeCall( hipMemcpy(A2p127, MRG32k3a::A2p127_host, sizeof(double) * 9, hipMemcpyHostToDevice ) );
			
			cutilSafeCall( hipMalloc((void**) &A1_pows, sizeof(double) * 11 * 3 * 3) );
			cutilSafeCall( hipMemcpy(A1_pows, MRG32k3a::A1_pows_host, sizeof(double) * 11 * 3 * 3, hipMemcpyHostToDevice ) );
			cutilSafeCall( hipMalloc((void**) &A2_pows, sizeof(double) * 11 * 3 * 3) );
			cutilSafeCall( hipMemcpy(A2_pows, MRG32k3a::A2_pows_host, sizeof(double) * 11 * 3 * 3, hipMemcpyHostToDevice ) );
}

__host__
void ParameterizedStatus<MRG32k3a::MRG32k3a>::release() {

	cutilSafeCall( hipFree(A2_pows) );
	cutilSafeCall( hipFree(A1_pows) );
	cutilSafeCall( hipFree(A2p127) );
	cutilSafeCall( hipFree(A1p127) );
	cutilSafeCall( hipFree(A2p76) );
	cutilSafeCall( hipFree(A1p76) );
	cutilSafeCall( hipFree(allStreams_) );
	
	delete [] allStreams_host;
}