	/***********************************************************************\
	* \file Created by jH@CKtheRipper, 04/04/11
	*       Stream and Substream classes declarations
	*       Inspired from Pierre L'Ecuyer's original RngStream.h
	* \author jH@CKtheRipper
	\***********************************************************************/


#include <hip/hip_runtime.h>

#include "Stream.h"
#include "ParameterizedStatus.h"
#include "utils.h" // TODO move to details

namespace shoverand {
	namespace prng {
		namespace MRG32k3a {

			/**
			The default seed of the package; will be the seed of the first
			declared RngStream, unless SetPackageSeed is called.
			*/
			double Stream::nextSeed__[6] =
			{
				12345.0, 12345.0, 12345.0, 12345.0, 12345.0, 12345.0
			};


			/**
			* Stream constructor
			*/
			__host__
			Stream::Stream () {

				/* Information on a stream. The arrays {Cg, Bg, Ig} contain respecively
						the current state of the stream, the starting state of the current
						SubStream, and the starting state of the stream. nextSeed__
						will be the seed of the next declared RngStream. */


				for (int i = 0; i < 6; ++i) {
					Cg_[i] = nextSeed__[i];
				}

				shoverand::utils::Math::MatVecModM (ParameterizedStatusMRG32k3a::A1p127_host, nextSeed__, nextSeed__, m1);
				shoverand::utils::Math::MatVecModM (ParameterizedStatusMRG32k3a::A2p127_host, &nextSeed__[3], &nextSeed__[3], m2);

			}

		} // end of namespace MRG32k3a
	} // end of namespace prng
} // end of namespace shoverand
