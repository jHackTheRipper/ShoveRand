#include "hip/hip_runtime.h"

// constant data for swaption portfolio: stored in device memory,
// initialised by host and read by device threads

__constant__ int    N, Nmat, Nopt, maturities[NOPT]; 
__constant__ float  delta, lambda[NN], L0[NN], swaprates[NOPT];


/* Monte Carlo LIBOR path calculation */

__device__ void path_calc(float *z, float *L)
{
  int   i, n;
  float sqez, lam, con1, v, vrat;

  for(n=0; n<Nmat; n++) {
    sqez = sqrtf(delta)*(*z);
    z += blockDim.x;  // note increment by blockDim.x, not 1
    v = 0.0f;

    for (i=n+1; i<N; i++) {
      lam  = lambda[i-n-1];
      con1 = delta*lam;
      v   += con1*L[i]/(1.0f+delta*L[i]);
      vrat = expf(con1*v + lam*(sqez-0.5f*con1));
      L[i] = L[i]*vrat;
    }
  }
}


/* forward path calculation storing data
   for subsequent reverse path calculation */

__device__ void path_calc_b1(float *z, float *L, float *L2)
{
  int   i, n;
  float sqez, lam, con1, v, vrat;

  for (i=0; i<N; i++) L2[i] = L[i];
   
  for(n=0; n<Nmat; n++) {
    sqez = sqrtf(delta)*(*z);
    z += blockDim.x;  // note increment by blockDim.x, not 1
    v = 0.0f;

    for (i=n+1; i<N; i++) {
      lam  = lambda[i-n-1];
      con1 = delta*lam;
      v   += con1*L[i]/(1.0f+delta*L[i]);
      vrat = expf(con1*v + lam*(sqez-0.5f*con1));
      L[i] = L[i]*vrat;

      // store these values for reverse path //
      L2[i+(n+1)*N] = L[i];
    }
  }
}


/* reverse path calculation of deltas using stored data */

__device__ void path_calc_b2(float *L_b, float *L2)
{
  int   i, n;
  float faci, v1;

  for (n=Nmat-1; n>=0; n--) {
    v1 = 0.0f;
    for (i=N-1; i>n; i--) {
      v1    += lambda[i-n-1]*L2[i+(n+1)*N]*L_b[i];
      faci   = delta/(1.0f+delta*L2[i+n*N]);
      L_b[i] = L_b[i]*L2[i+(n+1)*N]/L2[i+n*N]
              + v1*lambda[i-n-1]*faci*faci;
    }
  }
}


/* calculate the portfolio value v */

__device__ float portfolio(float *L)
{
  int   n, m, i;
  float v, b, s, swapval;
  float B[NMAT], S[NMAT];

  b = 1.0f;
  s = 0.0f;

  for (n=Nmat; n<N; n++) {
    b = b/(1.0f+delta*L[n]);
    s = s + delta*b;
    B[n-Nmat] = b;
    S[n-Nmat] = s;
  }

  v = 0.0f;

  for (i=0; i<Nopt; i++){
    m = maturities[i] -1;
    swapval = B[m] + swaprates[i]*S[m] - 1.0f;
    if (swapval < 0.0f)
      v += -100.0f*swapval;
  }

  // apply discount //

  for (n=0; n<Nmat; n++)
    v = v/(1.0f+delta*L[n]);

  return v;
}


/* calculate the portfolio value v, and its sensitivity to L */
/* hand-coded reverse mode sensitivity */

__device__ float portfolio_b(float *L, float *L_b) 
{
  int   m, n;
  float v, b, s, swapval;
  float B[NMAT], S[NMAT], B_b[NMAT], S_b[NMAT];

  b = 1.0f;
  s = 0.0f;
  for (m=0; m<N-Nmat; m++) {
    n    = m + Nmat;
    b    = b/(1.0f+delta*L[n]);
    s    = s + delta*b;
    B[m] = b;
    S[m] = s;
  }

  v = 0.0;

  for (m=0; m<N-Nmat; m++) {
    B_b[m] = 0.0f;
    S_b[m] = 0.0f;
  }

  for (n=0; n<Nopt; n++){
    m = maturities[n] - 1;
    swapval = B[m] + swaprates[n]*S[m] - 1.0f;
    if (swapval < 0.0f) {
      v      += -100.0f*swapval;
      S_b[m] += -100.0f*swaprates[n];
      B_b[m] += -100.0f;
    }
  }

  for (m=N-Nmat-1; m>=0; m--) {
    n = m + Nmat;
    B_b[m] += delta*S_b[m];
    L_b[n]  = -B_b[m]*B[m]*delta/(1.0f+delta*L[n]);
    if (m>0) {
      S_b[m-1] += S_b[m];
      B_b[m-1] += B_b[m]/(1.0f+delta*L[n]);
    }
  }

  // apply discount //

  b = 1.0f;
  for (n=0; n<Nmat; n++) b = b/(1.0f+delta*L[n]);

  v = b*v;

  for (n=0; n<Nmat; n++){
    L_b[n] = -v*delta/(1.0f+delta*L[n]);
  }

  for (n=Nmat; n<N; n++){
    L_b[n] = b*L_b[n];
  }

  return v;
}


/* -------------------------------------------------------- */

__global__ void Pathcalc_Portfolio_GPU(float *d_z, float *d_v, float *d_Lb)
{
  int   tid = threadIdx.x + blockIdx.x*blockDim.x;
  float L[NN], L2[L2_SIZE];
  float *L_b = L;

  /* initialise forward rates */
  for (int i=0; i<N; i++) L[i] = L0[i];

  /* move pointer to correct starting point */
  d_z = d_z + threadIdx.x + Nmat*blockIdx.x*blockDim.x;

  /* LIBOR path calculation and adjoint for Greeks */
  path_calc_b1(d_z,L,L2);
  d_v[tid] = portfolio_b(L,L_b);
  path_calc_b2(L_b, L2);
  d_Lb[tid] = L_b[NN-1];
}


__global__ void Pathcalc_Portfolio_GPU2(float *d_z, float *d_v)
{
  int   tid = threadIdx.x + blockIdx.x*blockDim.x;
  float L[NN];

  /* initialise forward rates */
  for (int i=0; i<N; i++) L[i] = L0[i];

  /* move pointer to correct starting point */
  d_z = d_z + threadIdx.x + Nmat*blockIdx.x*blockDim.x;

  /* LIBOR path calculation */
  path_calc(d_z,L);
  d_v[tid] = portfolio(L);
}
