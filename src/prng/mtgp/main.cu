#include "hip/hip_runtime.h"
/*
 *  main.cpp
 *  
 *
 *  Created by Jonathan PASSERAT-PALMBACH on 12/7/10.
 *  Copyright 2010 ISIMA/LIMOS. All rights reserved.
 *
 */


//#include <RNG.hxx>
//#include <MTGP.hxx>




extern "C" {
   #include <stdio.h>
   #include <stdlib.h>
   #include <errno.h>
}

#include <shoverand/util/myCutil.h>
#include "mtgp-cuda-common.h"
#include "mtgp32-cuda-cuterandAPI.h"



#define DATA_SIZE LARGE_SIZE

//typedef RNG<int, MTGP> MyRNG;

/*
__global__ void testShoveRand(uint32_t* ddata, MyRNG* rngTab) {
   
   MyRNG rng; // = rngTab + threadIdx.x;
   
   //rng->next(threadIdx.x);
   ddata[threadIdx.x] = rng.foo();
   
//   ddata[blockDim.x * blockIdx.x + threadIdx.x] = status[threadIdx.x + blockDim.x*blockIdx.x];
//   ddata[threadIdx.x] = status[threadIdx.x];
}*/


__global__ void testMTGP(uint32_t* ddata, mtgp32_kernel_status_t* dstatus) {
   mtgp32_init_shared_memory(dstatus);
   __syncthreads();

	for (int i = 0; i < 100; ++i)   mtgp32_uint32_device();
   
   ddata[blockDim.x * blockIdx.x + threadIdx.x] = mtgp32_uint32_device();
//   ddata[blockDim.x * blockIdx.x + threadIdx.x] = status[threadIdx.x + blockDim.x*blockIdx.x];
//   ddata[threadIdx.x] = status[threadIdx.x];

   __syncthreads();
}


int main(int argc, char *argv[])
{
   // LARGE_SIZE is a multiple of 16
   int num_data = 10000000;
   int block_num = 200;
   int num_unit;
   int r;
   mtgp32_kernel_status_t *d_status;
   uint32_t* d_texture[3];
   uint32_t* d_data;
   unsigned int timer = 0;
   uint32_t* h_data;
   hipError_t e;
   float gputime;
   
   CUT_DEVICE_INIT(argc, argv);
   
   myCutilSafeCall(hipMalloc((void**)&d_status,
                             sizeof(mtgp32_kernel_status_t) * block_num));
   myCutilSafeCall(hipMalloc((void**)&d_texture[0],
                             sizeof(uint32_t) * block_num * TBL_SIZE));
   myCutilSafeCall(hipMalloc((void**)&d_texture[1],
                             sizeof(uint32_t) * block_num * TBL_SIZE));
   myCutilSafeCall(hipMalloc((void**)&d_texture[2],
                             sizeof(uint32_t) * block_num * TBL_SIZE));

   make_constant(MTGPDC_PARAM_TABLE, block_num);
  // make_texture(MTGPDC_PARAM_TABLE, d_texture, block_num);
   make_kernel_data(d_status, MTGPDC_PARAM_TABLE, block_num);
   
   


    printf("generating 32-bit unsigned random numbers.\n");
    myCutilSafeCall(hipMalloc((void**)&d_data, sizeof(uint32_t) * block_num * THREAD_NUM));
    myCutilSafeCall(hipMemset(d_data, 0, block_num * THREAD_NUM));
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    h_data = (uint32_t *) malloc(sizeof(uint32_t) * block_num * THREAD_NUM);
    if (h_data == NULL) {
    printf("failure in allocating host memory for output data.\n");
    exit(1);
    }
    CUT_SAFE_CALL(cutStartTimer(timer));
    if (hipGetLastError() != hipSuccess) {
	printf("error has been occured before kernel call.\n");
	exit(1);
    }

/*
 	MyRNG rng_host;
   MyRNG* rng_device;
   
   rng_host.init();
   
   myCutilSafeCall(hipMalloc((void**)&rng_device, sizeof(MyRNG) * THREAD_NUM));
   testShoveRand<<< block_num, THREAD_NUM >>>(d_data, rng_device);
*/

    /* kernel call */
//    testMTGP<<< block_num, THREAD_NUM >>>(d_data, d_status);
    
    
   // hipDeviceSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
	printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
	exit(1);
    }
    CUT_SAFE_CALL(cutStopTimer(timer));
    myCutilSafeCall(
	hipMemcpy(h_data,
		   d_data,
		   sizeof(uint32_t) * block_num * THREAD_NUM,
		   hipMemcpyDeviceToHost));
    gputime = cutGetTimerValue(timer);


    for (int i = 0; i < block_num * THREAD_NUM; ++i) {
       printf ("h_data[%d] = %u\n", i, h_data[i]);
    }
      
   
    printf("generated numbers: %d\n", THREAD_NUM * block_num);
    printf("Processing time: %f (ms)\n", gputime);
    printf("Samples per second: %E \n", (THREAD_NUM * block_num) / (gputime * 0.001));
    CUT_SAFE_CALL(cutDeleteTimer(timer));
    //free memories
    free(h_data);
    myCutilSafeCall(hipFree(d_data));

   
   
   
   
   //finalize
   myCutilSafeCall(hipFree(d_status));
   myCutilSafeCall(hipFree(d_texture[0]));
   myCutilSafeCall(hipFree(d_texture[1]));
   myCutilSafeCall(hipFree(d_texture[2]));
#ifdef NEED_PROMPT
   CUT_EXIT(argc, argv);
#endif
}
