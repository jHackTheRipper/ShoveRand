/*
 * Sample Program for CUDA 2.3
 * written by M.Saito (saito@math.sci.hiroshima-u.ac.jp)
 *
 * This sample uses texture reference.
 * The generation speed of PRNG using texture is faster than using
 * constant tabel on Geforce GTX 260.
 *
 * MTGP32-11213
 * This program generates 32-bit unsigned integers.
 * The period of generated integers is 2<sup>23209</sup>-1.
 * This also generates single precision floating point numbers.
 */


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <stdint.h>
#include <inttypes.h>
#include <errno.h>
#include <stdlib.h>

#include "mtgp-cuda-common.h"

void printParams(mtgp32_params_fast_t* params);

/*
int get_suitable_block_num(int word_size, int thread_num, int large_size) {
    hipDeviceProp_t dev;
    hipDevice_t cuDevice;
    int max_thread_dev;
    int max_block, max_block_mem, max_block_dev;
    int major, minor, ver;

    CUDA_SAFE_CALL(hipGetDeviceProperties(&dev, 0));
    hipDeviceGet(&cuDevice, 0);
    hipDeviceComputeCapability(&major, &minor, cuDevice);
    max_block_mem = dev.sharedMemPerBlock / (large_size * word_size);
    if (major == 9999 && minor == 9999) {
	return -1;
    }
    ver = major * 100 + minor;
    if (ver <= 101) {
	max_thread_dev = 768;
    } else if (ver <= 103) {
	max_thread_dev = 1024;
    } else {
	max_thread_dev = 1024;
    }
    max_block_dev = max_thread_dev / thread_num;
    if (max_block_mem < max_block_dev) {
	max_block = max_block_mem;
    } else {
	max_block = max_block_dev;
    }
    return max_block * dev.multiProcessorCount;
}
*/

/**
 * This function initializes kernel I/O data.
 * @param d_status output kernel I/O data.
 * @param params MTGP32 parameters. needed for the initialization.
 */
void make_kernel_data(mtgp32_kernel_status_t *d_status,
                      mtgp32_params_fast_t params[],
                      int block_num) {
   mtgp32_kernel_status_t* h_status = (mtgp32_kernel_status_t *) malloc(block_num *
                                                                        sizeof(mtgp32_kernel_status_t));
   
   if (h_status == NULL) {
      printf("failure in allocating host memory for kernel I/O data.\n");
      exit(8);
   }
   for (int i = 0; i < block_num; i++) {
   
   //   printParams(&params[i]);
      mtgp32_init_state(&(h_status[i].status[0]), &params[i], i + 1);
   }
#if defined(DEBUG)
   
   printf("h_status[0].status[0]:%08x\n", h_status[0].status[0]);
   printf("h_status[0].status[1]:%08x\n", h_status[0].status[1]);
   printf("h_status[0].status[2]:%08x\n", h_status[0].status[2]);
   printf("h_status[0].status[3]:%08x\n\n", h_status[0].status[3]);
   
   printf("h_status[1].status[0]:%08x\n", h_status[1].status[0]);
   printf("h_status[1].status[1]:%08x\n", h_status[1].status[1]);
   printf("h_status[1].status[2]:%08x\n", h_status[1].status[2]);
   printf("h_status[1].status[3]:%08x\n\n", h_status[1].status[3]);
   
   printf("h_status[2].status[0]:%08x\n", h_status[2].status[0]);
   printf("h_status[2].status[1]:%08x\n", h_status[2].status[1]);
   printf("h_status[2].status[2]:%08x\n", h_status[2].status[2]);
   printf("h_status[2].status[3]:%08x\n\n", h_status[2].status[3]);
#endif
   CUDA_SAFE_CALL(hipMemcpy(d_status,
                             h_status,
                             sizeof(mtgp32_kernel_status_t) * block_num,
                             hipMemcpyHostToDevice));
   free(h_status);
}


void printParams(mtgp32_params_fast_t* params) {

      printf ("Printing parameters:\n \
            mexp: %d\n\
            pos: %d\n\
            sh1: %d\n\
            sh2: %d\n", params->mexp, params->pos, params->sh1, params->sh2);
  
      for (int i = 0; i < 16; ++i)     printf ("tbl[%d]= %u\n", i, params->tbl[i]);
      for (int i = 0; i < 16; ++i)     printf ("tmp_tbl[%d]= %u\n", i, params->tmp_tbl[i]);
      for (int i = 0; i < 16; ++i)     printf ("flt_tmp_tbl[%d]= %u\n", i, params->flt_tmp_tbl[i]);
      printf("mask: %u\n", params->mask);
      for (int i = 0; i < 21; ++i)     printf ("poly_sha1[%d]= %d\n", i, params->poly_sha1[i]);
}
