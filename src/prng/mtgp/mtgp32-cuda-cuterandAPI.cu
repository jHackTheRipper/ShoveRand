/**
 * Sample Program for CUDA 2.3
 * written by M.Saito (saito@math.sci.hiroshima-u.ac.jp)
 *
 * This sample uses texture reference.
 * The generation speed of PRNG using texture is faster than using
 * constant tabel on Geforce GTX 260.
 *
 * MTGP32-11213
 * This program generates 32-bit unsigned integers.
 * The period of generated integers is 2<sup>11213</sup>-1.
 * This also generates single precision floating point numbers.
 */
#define __STDC_FORMAT_MACROS 1
#define __STDC_CONSTANT_MACROS 1
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <shoverand/util/myCutil.h>
#include <stdint.h>
#include <inttypes.h>
#include <errno.h>
#include <stdlib.h>

#include "mtgp32-cuda-cuterandAPI.h"

//#include "mtgp-cuda-tex.h"
#include "mtgp-cuda-common.h"
#include "mtgp32-cuda-hostCalled.h"
extern "C" {
   #include "mtgp32-fast.h"
}




/********************************************************/
/********** begin of intialization functions ************/
/********************************************************/

/**
 * This function sets constants in device memory.
 * @param[in] params input, MTGP32 parameters.
 */
void make_constant(const mtgp32_params_fast_t params[],
    int block_num) {
    const int size1 = sizeof(uint32_t) * block_num;
    const int size2 = sizeof(uint32_t) * block_num * TBL_SIZE;
    uint32_t *h_pos_tbl;
    uint32_t *h_sh1_tbl;
    uint32_t *h_sh2_tbl;
    uint32_t *h_param_tbl;
    uint32_t *h_temper_tbl;
    uint32_t *h_single_temper_tbl;
#if 0
    uint32_t *h_mask;
#endif
    h_pos_tbl = (uint32_t *)malloc(size1);
    h_sh1_tbl = (uint32_t *)malloc(size1);
    h_sh2_tbl = (uint32_t *)malloc(size1);
    h_param_tbl = (uint32_t *)malloc(size2);
    h_temper_tbl = (uint32_t *)malloc(size2);
    h_single_temper_tbl = (uint32_t *)malloc(size2);
#if 0
    h_mask = (uint32_t *)malloc(sizeof(uint32_t));
#endif
    if (h_pos_tbl == NULL
	|| h_sh1_tbl == NULL
	|| h_sh2_tbl == NULL
	|| h_param_tbl == NULL
	|| h_temper_tbl == NULL
	|| h_single_temper_tbl == NULL
#if 0
	|| h_mask == NULL
#endif
	) {
	printf("failure in allocating host memory for constant table.\n");
	exit(1);
    }
#if 0
    h_mask = params[0].mask;
#endif
    for (int i = 0; i < block_num; i++) {
	h_pos_tbl[i] = params[i].pos;
	h_sh1_tbl[i] = params[i].sh1;
	h_sh2_tbl[i] = params[i].sh2;
	for (int j = 0; j < TBL_SIZE; j++) {
	    h_param_tbl[i * TBL_SIZE + j] = params[i].tbl[j];
	    h_temper_tbl[i * TBL_SIZE + j] = params[i].tmp_tbl[j];
	    h_single_temper_tbl[i * TBL_SIZE + j] = params[i].flt_tmp_tbl[j];
	}
    }
    // copy from malloc area only
    myCutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(pos_tbl), h_pos_tbl, size1));
    myCutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(sh1_tbl), h_sh1_tbl, size1));
    myCutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(sh2_tbl), h_sh2_tbl, size1));
    myCutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(param_tbl), h_param_tbl, size2));
    myCutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(temper_tbl), h_temper_tbl, size2));
    myCutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(single_temper_tbl),
				      h_single_temper_tbl, size2));
#if 0
    myCutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(&mask),
				      &h_mask, sizeof(uint32_t)));
#endif
    free(h_pos_tbl);
    free(h_sh1_tbl);
    free(h_sh2_tbl);
    free(h_param_tbl);
    free(h_temper_tbl);
    free(h_single_temper_tbl);
#if 0
    free(h_mask);
#endif
}


/**
 * This function sets texture lookup table.
 * @param params input, MTGP32 parameters.
 * @param d_texture_tbl device memory used for texture bind
 * @param block_num block number used for kernel call
 */
/*void make_texture(const mtgp32_params_fast_t params[],
                  uint32_t *d_texture_tbl[3],
                  int block_num) {
   const int count = block_num * TBL_SIZE;
   const int size = sizeof(uint32_t) * count;
   uint32_t *h_texture_tbl[3];
   int i, j;
   for (i = 0; i < 3; i++) {
      h_texture_tbl[i] = (uint32_t *)malloc(size);
      if (h_texture_tbl[i] == NULL) {
         for (j = 0; j < i; j++) {
            free(h_texture_tbl[i]);
         }
         printf("failure in allocating host memory for constant table.\n");
         exit(1);
      }
   }
   for (int i = 0; i < block_num; i++) {
      for (int j = 0; j < TBL_SIZE; j++) {
         h_texture_tbl[0][i * TBL_SIZE + j] = params[i].tbl[j];
         h_texture_tbl[1][i * TBL_SIZE + j] = params[i].tmp_tbl[j];
         h_texture_tbl[2][i * TBL_SIZE + j] = params[i].flt_tmp_tbl[j];
      }
   }
   myCutilSafeCall(hipMemcpy(d_texture_tbl[0], h_texture_tbl[0], size,
                             hipMemcpyHostToDevice));
   myCutilSafeCall(hipMemcpy(d_texture_tbl[1], h_texture_tbl[1], size,
                             hipMemcpyHostToDevice));
   myCutilSafeCall(hipMemcpy(d_texture_tbl[2], h_texture_tbl[2], size,
                             hipMemcpyHostToDevice));
   tex_param_ref.filterMode = hipFilterModePoint;
   tex_temper_ref.filterMode = hipFilterModePoint;
   tex_single_ref.filterMode = hipFilterModePoint;
   myCutilSafeCall(hipBindTexture(0, tex_param_ref, d_texture_tbl[0], size));
   myCutilSafeCall(hipBindTexture(0, tex_temper_ref, d_texture_tbl[1], size));
   myCutilSafeCall(hipBindTexture(0, tex_single_ref, d_texture_tbl[2], size));
   free(h_texture_tbl[0]);
   free(h_texture_tbl[1]);
   free(h_texture_tbl[2]);
}
*/
/********************************************************/
/************ end of intialization functions ************/
/********************************************************/


/********************************************************/
/************************ DEVICE API ********************/
/********************************************************/


