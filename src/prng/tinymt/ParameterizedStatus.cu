
#include "ParameterizedStatus.h"
#include "Stream.h"

#include <hip/hip_runtime.h>
#include <shoverand/util/myCutil.h>

#include <shoverand/util/tinymt/util.hxx>

// TODO let users defining it at runtime
const std::string ParameterizedStatus<shoverand::prng::TinyMT::TinyMT> :: filename_ = "../../test/tinymt32dc.0.20.txt";

__host__
void ParameterizedStatus<shoverand::prng::TinyMT::TinyMT> :: setUp (unsigned short inBlockId) {
	// TODO figure out how to statically determine template param instead of plain
	// unsigned int
	shoverand::util::TinyMT<unsigned int>::setParams(
			ParameterizedStatus<shoverand::prng::TinyMT::TinyMT> :: filename_.c_str(),
			&mat1_,
			&mat2_,
			&tmat_,
			inBlockId
	);

}

__host__
void ParameterizedStatus<shoverand::prng::TinyMT::TinyMT> :: shutdown() {

}
