#include "hip/hip_runtime.h"
#include <shoverand/prng/mrg32k3a/MRG32k3a.hxx>
#include <shoverand/prng/tinymt/TinyMT.hxx>
#include <shoverand/core/RNG.hxx>

#include <shoverand/distribution/variate_generator.hpp>
#include <shoverand/distribution/uniform_01.hpp>

#include <cstdlib>

#include <hip/hip_runtime.h>
#include <shoverand/util/myCutil.h>

#include <iostream> // debug purposes


// shortcuts :)
using shoverand::RNG;
using shoverand::MRG32k3a;
using shoverand::TinyMT;

typedef RNG < double, MRG32k3a > rng_type;


/** Kernel testing PRNG implementations */
__global__ void testPRNG(double* ddata, int inNbDataPerThread) {

	// this call might not work with devices of
	// compute capability < 2.x
	rng_type 	rng;

	// TODO old devices compliant version

	for (int i = 0; i < inNbDataPerThread; ++i) {
		ddata[inNbDataPerThread * (blockDim.x * blockIdx.x + threadIdx.x) + i] = rng.next();
	}
}


/** Kernel testing variate_generator facility */
//__global__ void testVariateGenerator(double* ddata,  ParameterizedStatusType* param) {
__global__ void testVariateGenerator(double* ddata) {

	// this call might not work with devices of
	// compute capability < 2.x
	typedef rng_type				    				 randomengine;
	typedef shoverand::uniform_01< float, float>         distribution;

	randomengine 	rng;

	distribution myDistribution;

	shoverand::variate_generator
	<
	randomengine,
	distribution
	>
	myVariateGenerator(rng, myDistribution);

	ddata[blockDim.x * blockIdx.x + threadIdx.x] = myVariateGenerator();
}


int main(int, char **) {

	size_t memFree;
	size_t memTotal;

	hipMemGetInfo(&memFree, &memTotal);
	std :: cerr << "Available device memory at the beginning: " << memFree << "/" << memTotal << std::endl;

	int nbDataPerThread = 100;
	int blockNum = 512;
	int threadNum = 512;
	int nbData = nbDataPerThread * blockNum * threadNum;
	int dataSize = nbData * sizeof(double);

	double* d_data;
	double* h_data;
	hipError_t e;
	float gputime;


	// create timers
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	// allocate memory for data on device
	myCutilSafeCall( hipMalloc((void**) &d_data, dataSize) );
	myCutilSafeCall( hipMemset(d_data, 0, dataSize) );


	if (hipGetLastError() != hipSuccess) {
		std::cerr << "error has occured before kernel call." << std::endl;
		exit(1);
	}

	// init step from host side
	rng_type ::init(blockNum);

	hipEventRecord(start, 0);


	// --- kernel calls ---
	//	testVariateGenerator<<< block_num, thread_num >>>(d_data);
	testPRNG<<< blockNum, threadNum >>>(d_data, nbDataPerThread);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	e = hipGetLastError();
	if (e != hipSuccess) {
		std::cerr << "failure in kernel call.\n" << hipGetErrorString(e) << std::endl;
		exit(2);
	}

	// allocate memory to get results back on the host
	h_data = new double[dataSize];

	if (h_data == NULL) {
		std::cerr << "failure in allocating host memory for output data." << std::endl;
		exit(3);
	}

	myCutilSafeCall(
			hipMemcpy(h_data,
					d_data,
					dataSize,
					hipMemcpyDeviceToHost));
	hipEventElapsedTime(&gputime, start, stop);


	for (int i = 0; i < nbData; ++i) {
		std::cout << "h_data[" << i << "] = " << h_data[i] << std::endl;
	}


	std::cout << "generated numbers: " << nbData << std::endl;
	std::cout << "Processing time: " << gputime << " (ms)" << std::endl;
	std::cout << "Samples per second: " << nbData / (gputime * 0.001) << std::endl;

	//free memory
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// release memory allocated on device from host
	rng_type :: release();

	delete [] h_data;
	myCutilSafeCall(hipFree(d_data));

	hipMemGetInfo(&memFree, &memTotal);
	std :: cerr << "Available device memory at the end: " << memFree << "/" << memTotal << std::endl;
}
