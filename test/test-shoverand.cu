#include "hip/hip_runtime.h"
#include <shoverand/prng/mrg32k3a/MRG32k3a.hxx>
#include <shoverand/prng/tinymt/TinyMT.hxx>
#include <shoverand/core/RNG.hxx>

#include <shoverand/distribution/variate_generator.hpp>
#include <shoverand/distribution/uniform_01.hpp>

#include <cstdlib>

#include <hip/hip_runtime.h>
#include <shoverand/util/myCutil.h>

#include <iostream> // debug purposes


// shortcuts :)
using shoverand::RNG;
using shoverand::MRG32k3a;
using shoverand::TinyMT;

typedef RNG < float, MRG32k3a > rng_type;


/** Kernel testing PRNG implementations */
__global__ void testPRNG(double* ddata) {

	// this call might not work with devices of
	// compute capability < 2.x
	rng_type 	rng;

	// TODO old devices compliant version

	ddata[blockDim.x * blockIdx.x + threadIdx.x] = rng.next();
	__syncthreads();
}


/** Kernel testing variate_generator facility */
//__global__ void testVariateGenerator(double* ddata,  ParameterizedStatusType* param) {
__global__ void testVariateGenerator(double* ddata) {

	// this call might not work with devices of
	// compute capability < 2.x
	typedef rng_type				    				 randomengine;
	typedef shoverand::uniform_01< float, float>         distribution;

	randomengine 	rng;

	distribution myDistribution;

	shoverand::variate_generator
	<
	randomengine,
	distribution
	>
	myVariateGenerator(rng, myDistribution);

	ddata[blockDim.x * blockIdx.x + threadIdx.x] = myVariateGenerator();
}


int main(int, char **) {

	size_t memFree;
	size_t memTotal;

	hipMemGetInfo(&memFree, &memTotal);
	std :: cerr << "Available device memory at the beginning: " << memFree << "/" << memTotal << std::endl;

	int block_num = 3;
	int thread_num = 512;
	int data_size = block_num * thread_num * sizeof(double);

	double* d_data;
	double* h_data;
	hipError_t e;
	float gputime;


	// create timers
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	// allocate memory for data on device
	myCutilSafeCall( hipMalloc((void**) &d_data, data_size) );
	myCutilSafeCall( hipMemset(d_data, 0, data_size) );


	if (hipGetLastError() != hipSuccess) {
		std::cerr << "error has occured before kernel call." << std::endl;
		exit(1);
	}

	// init step from host side
	rng_type ::init(block_num);

	hipEventRecord(start, 0);


	// --- kernel calls ---
	//	testVariateGenerator<<< block_num, thread_num >>>(d_data);
	testPRNG<<< block_num, thread_num >>>(d_data);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	e = hipGetLastError();
	if (e != hipSuccess) {
		std::cerr << "failure in kernel call.\n" << hipGetErrorString(e) << std::endl;
		exit(2);
	}

	// allocate memory to get results back on the host
	h_data = new double[data_size];

	if (h_data == NULL) {
		std::cerr << "failure in allocating host memory for output data." << std::endl;
		exit(3);
	}

	myCutilSafeCall(
			hipMemcpy(h_data,
					d_data,
					data_size,
					hipMemcpyDeviceToHost));
	hipEventElapsedTime(&gputime, start, stop);


	for (int i = 0; i < block_num * thread_num; ++i) {
		std::cout << "h_data[" << i << "] = " << h_data[i] << std::endl;
	}


	std::cout << "generated numbers: " << thread_num * block_num << std::endl;
	std::cout << "Processing time: " << gputime << " (ms)" << std::endl;
	std::cout << "Samples per second: " << (thread_num * block_num) / (gputime * 0.001) << std::endl;

	//free memory
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// release memory allocated on device from host
	rng_type :: release();

	delete [] h_data;
	myCutilSafeCall(hipFree(d_data));

	hipMemGetInfo(&memFree, &memTotal);
	std :: cerr << "Available device memory at the end: " << memFree << "/" << memTotal << std::endl;
}
