#include "hip/hip_runtime.h"
#include <shoverand/prng/mrg32k3a/ParameterizedStatus.h>
#include <shoverand/prng/mrg32k3a/MRG32k3a.hxx>
#include <shoverand/core/RNG.hxx>

#include <cstdlib>

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cutil_inline_runtime.h>


#include <iostream> // debug purposes

// shortcuts :)
using shoverand::RNG;
using shoverand::MRG32k3a;
typedef RNG< float, MRG32k3a > ::ParameterizedStatusType ParameterizedStatusType;



__global__ void testMRG32k3a(double* ddata,  ParameterizedStatusType* param) {

	// this call might not work with devices of
	// compute capability < 2.x
	RNG < float, MRG32k3a > 	rng(param);
	rng.init();

   // TODO old devices compliant version
   
	ddata[blockDim.x * blockIdx.x + threadIdx.x] = rng.next();
   __syncthreads();
}


int main(int, char **) {

   int block_num = 3;
   int thread_num = 512;
   int data_size = block_num * thread_num * sizeof(double);
   
   double* d_data;
   double* h_data;
   hipError_t e;
   float gputime;
   
   // create timers 
   hipEvent_t start;
   hipEvent_t stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   
   
   // allocate memory for data on device
   cutilSafeCall( hipMalloc((void**) &d_data, data_size) );
   cutilSafeCall( hipMemset(d_data, 0, data_size) );


	// ParameterizedStatus initialization on both sides
	ParameterizedStatusType* 	 status_host = new ParameterizedStatusType(); // TODO maybe change to builder method
	status_host->setUp(block_num);

   ParameterizedStatusType*    status_device;
   cutilSafeCall( hipMalloc((void**) &status_device, sizeof(ParameterizedStatusType)) );  
   cutilSafeCall( hipMemcpy(status_device, status_host, sizeof(ParameterizedStatusType), hipMemcpyHostToDevice) );


   if (hipGetLastError() != hipSuccess) {
      std::cerr << "error has occured before kernel call." << std::endl;
      exit(1);
   }
   
   hipEventRecord(start, 0);
   

   // kernel call
   testMRG32k3a<<< block_num, thread_num >>>(d_data, status_device);
   
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   
   e = hipGetLastError();
   if (e != hipSuccess) {
      std::cerr << "failure in kernel call.\n" << hipGetErrorString(e) << std::endl;
      exit(2);
   }
   
   // allocate memory to get results back on the host
   h_data = new double[data_size];
   
   if (h_data == NULL) {
      std::cerr << "failure in allocating host memory for output data." << std::endl;
      exit(3);
   }
   
   cutilSafeCall(
                 hipMemcpy(h_data,
                            d_data,
                            data_size,
                            hipMemcpyDeviceToHost));
   hipEventElapsedTime(&gputime, start, stop);
   
   
   for (int i = 0; i < block_num * thread_num; ++i) {
		std::cout << "h_data[" << i << "] = " << h_data[i] << std::endl;
   }
   
   
   std::cout << "generated numbers: " << thread_num * block_num << std::endl;
   std::cout << "Processing time: " << gputime << " (ms)" << std::endl;
   std::cout << "Samples per second: " << (thread_num * block_num) / (gputime * 0.001) << std::endl; 
   
   //free memory
   hipEventDestroy(start);
   hipEventDestroy(stop);
   
   delete [] h_data;
   cutilSafeCall(hipFree(d_data));

	cutilSafeCall(hipFree(status_device));
	delete status_host;
   
}
