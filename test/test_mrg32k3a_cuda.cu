#include "hip/hip_runtime.h"
// #include <Stream.cu> // already included by ParameterizedStatus.h
#include <ParameterizedStatus.h>
#include <SubStream.h>
#include <MRG32k3a.hxx>

#include <cstdlib>

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cutil_inline_runtime.h>


#include <iostream> // debug purposes

// shortcuts :)
using shoverand::prng::core::RNG;
typedef RNG< float, MRG32k3a::MRG32k3a > ::ParameterizedStatusType ParameterizedStatusType;



__global__ void testMRG32k3a(double* ddata,  ParameterizedStatusType* param) {

	// this call could not work with devices of
	// compute capability < 2.x
	//MRG32k3a::SubStream s(param);

	RNG < float, MRG32k3a::MRG32k3a > 	rng(param);
	rng.init();

   // old devices compliant version
//    MRG32k3a::SubStream* s = allSubStreams + (blockDim.x * blockIdx.x + threadIdx.x);
//    s->init(allStreams);
   
	ddata[blockDim.x * blockIdx.x + threadIdx.x] = rng.next(); // IT WORKS!!!!!
   __syncthreads();
}


int main(int, char **) {

   int block_num = 3;
   int thread_num = 512;
   int data_size = block_num * thread_num * sizeof(double);
   
   double* d_data;
   double* h_data;
   hipError_t e;
   float gputime;
   
   // create timers 
   hipEvent_t start;
   hipEvent_t stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   
   
   // allocate memory for data on device
   cutilSafeCall( hipMalloc((void**) &d_data, data_size) );
   cutilSafeCall( hipMemset(d_data, 0, data_size) );



	ParameterizedStatusType* 	 status_host = new MRG32k3a::ParameterizedStatusMRG32k3a(); // TODO change to builder method
	status_host->setUp(block_num);
   ParameterizedStatusType*    status_device;
   cutilSafeCall( hipMalloc((void**) &status_device, sizeof(ParameterizedStatusType)) );  
   cutilSafeCall( hipMemcpy(status_device, status_host, sizeof(ParameterizedStatusType), hipMemcpyHostToDevice) );


   if (hipGetLastError() != hipSuccess) {
      std::cerr << "error has occured before kernel call." << std::endl;
      exit(1);
   }
   
   hipEventRecord(start, 0);
   

   // kernel call
   testMRG32k3a<<< block_num, thread_num >>>(d_data, status_device);
   
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   
   e = hipGetLastError();
   if (e != hipSuccess) {
      std::cerr << "failure in kernel call.\n" << hipGetErrorString(e) << std::endl;
      exit(2);
   }
   
   // allocate memory to get results back on the host
   h_data = new double[data_size];
   
   if (h_data == NULL) {
      std::cerr << "failure in allocating host memory for output data." << std::endl;
      exit(3);
   }
   
   cutilSafeCall(
                 hipMemcpy(h_data,
                            d_data,
                            data_size,
                            hipMemcpyDeviceToHost));
   hipEventElapsedTime(&gputime, start, stop);
   
   
   for (int i = 0; i < block_num * thread_num; ++i) {
		std::cout << "h_data[" << i << "] = " << h_data[i] << std::endl;
   }
   
   
   std::cout << "generated numbers: " << thread_num * block_num << std::endl;
   std::cout << "Processing time: " << gputime << " (ms)" << std::endl;
   std::cout << "Samples per second: " << (thread_num * block_num) / (gputime * 0.001) << std::endl; 
   
   //free memories
   hipEventDestroy(start);
   hipEventDestroy(stop);
   
   delete [] h_data;
   cutilSafeCall(hipFree(d_data));

	cutilSafeCall(hipFree(status_device));
	delete status_host;
   
}
