#include "hip/hip_runtime.h"
#include <Stream.cu>
#include <ParameterizedStatus.h>
#include <SubStream.h>

#include <MRG32k3a.hxx>

#include <cstdio>
#include <cstdlib>
#include <cerrno>


#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cutil_inline_runtime.h>


// shortcut :)
typedef RNG< float, MRG32k3a::MRG32k3a > ::ParameterizedStatusType ParameterizedStatusType;


__global__ void testMRG32k3a(double* ddata,  ParameterizedStatusType* param) {

	// this call could not work with devices of
	// compute capability < 2.x
	//MRG32k3a::SubStream s(param);

	RNG < float, MRG32k3a::MRG32k3a > 	rng(param);
	rng.init();

   // old devices compliant version
//    MRG32k3a::SubStream* s = allSubStreams + (blockDim.x * blockIdx.x + threadIdx.x);
//    s->init(allStreams);
   
  // ddata[blockDim.x * blockIdx.x + threadIdx.x] = s.next(); // IT WORKS!!!!!
	ddata[blockDim.x * blockIdx.x + threadIdx.x] = rng.next(); // IT WORKS!!!!!
   __syncthreads();
}


int main(int, char **) {

   int block_num = 3;
   int thread_num = 512;
   int data_size = block_num * thread_num * sizeof(double);
   
   double* d_data;
   double* h_data;
   hipError_t e;
   float gputime;
   
   // create timers 
   hipEvent_t start;
   hipEvent_t stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   
   
   // allocate memory for data on device
   cutilSafeCall( hipMalloc((void**) &d_data, data_size) );
   cutilSafeCall( hipMemset(d_data, 0, data_size) );



	ParameterizedStatusType* 	 status_host = new MRG32k3a::ParameterizedStatusMRG32k3a(); // TODO change by builder method
	status_host->setUp(block_num);
   ParameterizedStatusType*    status_device;
   cutilSafeCall( hipMalloc((void**) &status_device, sizeof(ParameterizedStatusType)) );  
   cutilSafeCall( hipMemcpy(status_device, status_host, sizeof(ParameterizedStatusType), hipMemcpyHostToDevice) );


   if (hipGetLastError() != hipSuccess) {
      fprintf(stderr, "error has occured before kernel call.\n");
      exit(1);
   }
   
   hipEventRecord(start, 0);
   

   // kernel call
   testMRG32k3a<<< block_num, thread_num >>>(d_data, status_device);
   
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   
   e = hipGetLastError();
   if (e != hipSuccess) {
      fprintf(stderr, "failure in kernel call.\n%s\n", hipGetErrorString(e));
      exit(2);
   }
   
   // allocate memory to get results back on the host
   h_data = new double[data_size];
   
   if (h_data == NULL) {
      fprintf(stderr, "failure in allocating host memory for output data.\n");
      exit(3);
   }
   
   cutilSafeCall(
                 hipMemcpy(h_data,
                            d_data,
                            data_size,
                            hipMemcpyDeviceToHost));
   hipEventElapsedTime(&gputime, start, stop);
   
   
   for (int i = 0; i < block_num * thread_num; ++i) {
		std::cout << "h_data[" << i << "] = " << h_data[i] << std::endl;
   }
   
   
   printf("generated numbers: %d\n", thread_num * block_num);
   printf("Processing time: %f (ms)\n", gputime);
   printf("Samples per second: %E \n", (thread_num * block_num) / (gputime * 0.001));
   
   //free memories
   hipEventDestroy(start);
   hipEventDestroy(stop);
   
   delete [] h_data;
   cutilSafeCall(hipFree(d_data));

	cutilSafeCall(hipFree(status_device));
	delete status_host;
   
}
