#include "hip/hip_runtime.h"
//#include <shoverand/prng/mrg32k3a/ParameterizedStatus.h>
#include <shoverand/prng/mrg32k3a/MRG32k3a.hxx>
#include <shoverand/core/RNG.hxx>


#include <shoverand/prng/tinymt/utils.hxx>

#include <shoverand/distribution/variate_generator.hpp>
#include <shoverand/distribution/uniform_01.hpp>

#include <cstdlib>

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cutil_inline_runtime.h>


#include <iostream> // debug purposes


// shortcuts :)
using shoverand::RNG;
using shoverand::MRG32k3a;
//typedef RNG< float, MRG32k3a > ::ParameterizedStatusType ParameterizedStatusType;


/** Kernel testing MRG32k3a implementation */
//__global__ void testMRG32k3a(double* ddata,  ParameterizedStatusType* param) {
__global__ void testMRG32k3a(double* ddata) {

	// this call might not work with devices of
	// compute capability < 2.x
	RNG < float, MRG32k3a > 	rng;
	//rng.init();

   // TODO old devices compliant version
   
	ddata[blockDim.x * blockIdx.x + threadIdx.x] = rng.next();
   __syncthreads();
}


/** Kernel testing variate_generator facility */
//__global__ void testVariateGenerator(double* ddata,  ParameterizedStatusType* param) {
__global__ void testVariateGenerator(double* ddata) {

	// this call might not work with devices of
	// compute capability < 2.x
	typedef RNG < float, MRG32k3a >  				 randomengine;
	typedef boost::uniform_01< float, float>      distribution;

	randomengine 	rng;
	//rng.init();

	distribution myDistribution;

	boost::variate_generator
         <
         randomengine,
         distribution
         >
         myVariateGenerator(rng, myDistribution);
   
	ddata[blockDim.x * blockIdx.x + threadIdx.x] = myVariateGenerator();
}


int main(int, char **) {

   int block_num = 3;
   int thread_num = 512;
   int data_size = block_num * thread_num * sizeof(double);
   
   double* d_data;
   double* h_data;
   hipError_t e;
   float gputime;
   
   // create timers 
   hipEvent_t start;
   hipEvent_t stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   
   
   // allocate memory for data on device
   cutilSafeCall( hipMalloc((void**) &d_data, data_size) );
   cutilSafeCall( hipMemset(d_data, 0, data_size) );


   if (hipGetLastError() != hipSuccess) {
      std::cerr << "error has occured before kernel call." << std::endl;
      exit(1);
   }
   
	RNG< float, MRG32k3a > ::init(block_num);

   hipEventRecord(start, 0);
   

   // kernel call
   //testMRG32k3a<<< block_num, thread_num >>>(d_data, status_device);
	//testVariateGenerator<<< block_num, thread_num >>>(d_data, status_device);
	testMRG32k3a<<< block_num, thread_num >>>(d_data);
	//testVariateGenerator<<< block_num, thread_num >>>(d_data);
   
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   
   e = hipGetLastError();
   if (e != hipSuccess) {
      std::cerr << "failure in kernel call.\n" << hipGetErrorString(e) << std::endl;
      exit(2);
   }
   
   // allocate memory to get results back on the host
   h_data = new double[data_size];
   
   if (h_data == NULL) {
      std::cerr << "failure in allocating host memory for output data." << std::endl;
      exit(3);
   }
   
   cutilSafeCall(
                 hipMemcpy(h_data,
                            d_data,
                            data_size,
                            hipMemcpyDeviceToHost));
   hipEventElapsedTime(&gputime, start, stop);
   
   
   for (int i = 0; i < block_num * thread_num; ++i) {
		std::cout << "h_data[" << i << "] = " << h_data[i] << std::endl;
   }
   
   
   std::cout << "generated numbers: " << thread_num * block_num << std::endl;
   std::cout << "Processing time: " << gputime << " (ms)" << std::endl;
   std::cout << "Samples per second: " << (thread_num * block_num) / (gputime * 0.001) << std::endl; 
   
   //free memory
   hipEventDestroy(start);
   hipEventDestroy(stop);
   
	RNG< float, MRG32k3a > :: release();
   delete [] h_data;
   cutilSafeCall(hipFree(d_data));   
}
