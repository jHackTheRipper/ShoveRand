#include "hip/hip_runtime.h"
#include <shoverand/prng/mrg32k3a/MRG32k3a.hxx>
#include <shoverand/core/RNG.hxx>


#include <shoverand/prng/tinymt/utils.hxx>

#include <shoverand/distribution/variate_generator.hpp>
#include <shoverand/distribution/uniform_01.hpp>

#include <cstdlib>

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cutil_inline_runtime.h>


#include <iostream> // debug purposes

// shortcuts :)
using shoverand::RNG;
using shoverand::MRG32k3a;

__global__ void testMultiple1(float* ddata) {

	RNG < float, MRG32k3a > 	rng;
   
	ddata[blockDim.x * blockIdx.x + threadIdx.x] = rng.next();
}

__global__ void testMultiple2(float* ddata) {

	RNG < float, MRG32k3a > 	rng;

	ddata[blockDim.x * blockIdx.x + threadIdx.x] = rng.next();
}

__global__ void testMultiple3(float* ddata) {

	RNG < float, MRG32k3a > 	rng;

	ddata[blockDim.x * blockIdx.x + threadIdx.x] = rng.next();
}

void testMultipleInitializations() {

	size_t memFree;
	size_t memTotal;

	hipMemGetInfo(&memFree, &memTotal);
	std :: cerr << "Available device memory at the beginning: " << memFree << "/" << memTotal << std::endl;

   int block_num = 3;
   int thread_num = 512;
   int data_size = block_num * thread_num * sizeof(double);
   
   double* d_data;
   double* h_data;
   hipError_t e;
   float gputime;

   
   // create timers 
   hipEvent_t start;
   hipEvent_t stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   
   
   // allocate memory for data on device
   cutilSafeCall( hipMalloc((void**) &d_data, data_size) );
   cutilSafeCall( hipMemset(d_data, 0, data_size) );


   if (hipGetLastError() != hipSuccess) {
      std::cerr << "error has occured before kernel call." << std::endl;
      exit(1);
   }
   
	
   hipEventRecord(start, 0);
   

   float* d_bigdata;
	cutilSafeCall( hipMalloc( (void**) &d_bigdata, sizeof(float) * 1024*1024) );
	
	std::vector<int> paramVector;
	paramVector.push_back(20);
	paramVector.push_back(60);
	paramVector.push_back(10);

	RNG< float, MRG32k3a > ::init ( paramVector );
	//for (int i = 0; i < 10; ++i) {
	
		testMultiple1<<< 20, 64 >>>  ( d_bigdata );
		hipDeviceSynchronize();
// 		testMultiple2<<< 60, 128 >>> ( d_bigdata );
// 		hipDeviceSynchronize();
// 		testMultiple3<<< 10, 256 >>> ( d_bigdata );
// 		hipDeviceSynchronize();
//    }


   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   
   e = hipGetLastError();
   if (e != hipSuccess) {
      std::cerr << "failure in kernel call.\n" << hipGetErrorString(e) << std::endl;
      exit(2);
   }
   
   // allocate memory to get results back on the host
   h_data = new double[data_size];
   
   if (h_data == NULL) {
      std::cerr << "failure in allocating host memory for output data." << std::endl;
      exit(3);
   }
   
   cutilSafeCall(
                 hipMemcpy(h_data,
                            d_bigdata,
                            data_size,
                            hipMemcpyDeviceToHost));
   hipEventElapsedTime(&gputime, start, stop);
   
   
   for (int i = 0; i < block_num * thread_num; ++i) {
		std::cout << "h_data[" << i << "] = " << h_data[i] << std::endl;
   }

   std::cout << "generated numbers: " << thread_num * block_num << std::endl;
   std::cout << "Processing time: " << gputime << " (ms)" << std::endl;
   std::cout << "Samples per second: " << (thread_num * block_num) / (gputime * 0.001) << std::endl; 
   
   //free memory
   hipEventDestroy(start);
   hipEventDestroy(stop);

	RNG< float, MRG32k3a > :: release();
	
   delete [] h_data;
   cutilSafeCall(hipFree(d_data));   

	hipMemGetInfo(&memFree, &memTotal);
	std :: cerr << "Available device memory at the end: " << memFree << "/" << memTotal << std::endl;
}
