#include "hip/hip_runtime.h"
#include <shoverand/prng/mrg32k3a/MRG32k3a.hxx>
#include <shoverand/core/RNG.hxx>


#include <shoverand/prng/tinymt/utils.hxx>

#include <shoverand/distribution/variate_generator.hpp>
#include <shoverand/distribution/uniform_01.hpp>

#include <cstdlib>

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cutil_inline_runtime.h>

#include <algorithm> // std::accumulate

#include <iostream> // debug purposes

// shortcuts :)
using shoverand::RNG;
using shoverand::MRG32k3a;

__global__ void testMultiple1(float* ddata) {

	RNG < float, MRG32k3a > 	rng;

	ddata[blockDim.x * blockIdx.x + threadIdx.x] = rng.next();
}

__global__ void testMultiple2(float* ddata) {

	RNG < float, MRG32k3a > 	rng;

	ddata[blockDim.x * blockIdx.x + threadIdx.x] = rng.next();
}

__global__ void testMultiple3(float* ddata) {

	RNG < float, MRG32k3a > 	rng;

	ddata[blockDim.x * blockIdx.x + threadIdx.x] = rng.next();
}

void testMultipleInitializations() {

	size_t memFree;
	size_t memTotal;

	hipMemGetInfo(&memFree, &memTotal);
	std :: cerr << "Available device memory at the beginning: " << memFree << "/" << memTotal << std::endl;

	std::vector<int> blocksInKernels;
	blocksInKernels.push_back(20);
	blocksInKernels.push_back(60);
	blocksInKernels.push_back(10);

	std::vector<int> threadsPerBlocksInKernels;
	threadsPerBlocksInKernels.push_back(64);
	threadsPerBlocksInKernels.push_back(128);
	threadsPerBlocksInKernels.push_back(256);

	int block_num = std::accumulate ( blocksInKernels.begin(), blocksInKernels.end(), 0 );
	int thread_num = std::accumulate ( threadsPerBlocksInKernels.begin(), threadsPerBlocksInKernels.end(), 0 );
	int data_size = block_num * thread_num * sizeof(float);



	if (hipGetLastError() != hipSuccess) {
		std::cerr << "error has occurred before kernel call." << std::endl;
		exit(1);
	}



	// allocate memory to get results back on the host
	float* h_data = new float[1024*1024];
	if (h_data == NULL) {
		std::cerr << "failure in allocating host memory for output data." << std::endl;
		exit(3);
	}
	bzero(h_data, 1024*1024*sizeof(float));

	float* d_bigdata;
	cutilSafeCall( hipMalloc( (void**) &d_bigdata, sizeof(float) * 1024 * 1024) );
	cutilSafeCall( hipMemset( d_bigdata, 0, sizeof(float) * 1024 * 1024));



	RNG< float, MRG32k3a > ::init ( blocksInKernels );
	//for (int i = 0; i < 10; ++i) {

	testMultiple1<<< blocksInKernels[0], threadsPerBlocksInKernels[0] >>>  ( d_bigdata );
	hipError_t e = hipGetLastError();
	if (e != hipSuccess) {
		std::cerr << "failure in kernel call.\n" << hipGetErrorString(e) << std::endl;
		exit(2);
	}
	hipDeviceSynchronize();

	cutilSafeCall(
			hipMemcpy(h_data,
					d_bigdata,
					1024*1024,
					hipMemcpyDeviceToHost));

	for (int i = 0; i < blocksInKernels[0] * threadsPerBlocksInKernels[0]; ++i) {
		std::cout << "h_data[" << i << "] = " << h_data[i] << std::endl;
	}

	// 		testMultiple2<<< blocksInKernels[1], threadsPerBlocksInKernels[1] >>> ( d_bigdata );
	// 		hipDeviceSynchronize();
	// 		testMultiple3<<< blocksInKernels[2], threadsPerBlocksInKernels[2] >>> ( d_bigdata );
	// 		hipDeviceSynchronize();
	//    }




	// release memory
	RNG< float, MRG32k3a > :: release();
	delete [] h_data;
	cutilSafeCall(hipFree(d_bigdata));

	hipMemGetInfo(&memFree, &memTotal);
	std :: cerr << "Available device memory at the end: " << memFree << "/" << memTotal << std::endl;
}
